#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

#include <helper_functions.h>  // helper functions for string parsing
#include <hip/hip_runtime_api.h>  // helper functions CUDA error checking and initialization

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(float *h_CallResult, float *h_PutResult,
                                float *h_StockPrice, float *h_OptionStrike,
                                float *h_OptionYears, float Riskfree,
                                float Volatility, int optN);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;
const int NUM_ITERATIONS = 512;

const int OPT_SZ = OPT_N * sizeof(float);
const float RISKFREE = 0.02f;
const float VOLATILITY = 0.30f;

#define DIV_UP(a, b) (((a) + (b)-1) / (b))

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////

/*
 * DISCLAIMER: The following file has been slightly modified to ensure
 * compatibility with Clad and to serve as a Clad demo. Specifically, parts of
 * the original `main` function have been moved to a separate function to use
 * `clad::gradient` on. Furthermore, Clad cannot clone checkCudaErrors
 * successfully, so these calls have been omitted. The same applies to the
 * hipDeviceSynchronize function. New helper functions are included in another
 * file and invoked here to verify the gradient's results. Since Clad cannot
 * handle timers at the moment, the time measurement is included in
 * `main` and doesn't time exclusively the original kernel execution, but the
 * whole `launch` function and its gradient are timed in this version.
 *
 * The original file is available in NVIDIA's cuda-samples repository on GitHub.
 *
 * Relevant documentation regarding the problem at hand can be found in NVIDIA's
 * cuda-samples repository. Using Clad, we compute some of the Greeks
 * (sensitivities) for the Black-Scholes model and verify them against
 * approximations of their theoretical values as denoted in Wikipedia
 * (https://en.wikipedia.org/wiki/Black%E2%80%93Scholes_model).
 *
 * To build and run the demo, use the following command: make run
 */

#include "clad/Differentiator/Differentiator.h"
#include <helper_grad_verify.h>

void launch(float* h_CallResultCPU, float* h_CallResultGPU,
            float* h_PutResultCPU, float* h_PutResultGPU, float* h_StockPrice,
            float* h_OptionStrike, float* h_OptionYears) {

  //'d_' prefix - GPU (device) memory space
  float
      // Results calculated by GPU
      *d_CallResult = nullptr,
      *d_PutResult = nullptr,
      // GPU instance of input data
      *d_StockPrice = nullptr, *d_OptionStrike = nullptr,
      *d_OptionYears = nullptr;

  printf("...allocating GPU memory for options.\n");
  hipMalloc((void**)&d_CallResult, OPT_SZ);
  hipMalloc((void**)&d_PutResult, OPT_SZ);
  hipMalloc((void**)&d_StockPrice, OPT_SZ);
  hipMalloc((void**)&d_OptionStrike, OPT_SZ);
  hipMalloc((void**)&d_OptionYears, OPT_SZ);

  // Copy options data to GPU memory for further processing
  printf("...copying input data to GPU mem.\n");
  hipMemcpy(d_StockPrice, h_StockPrice, OPT_SZ, hipMemcpyHostToDevice);
  hipMemcpy(d_OptionStrike, h_OptionStrike, OPT_SZ, hipMemcpyHostToDevice);
  hipMemcpy(d_OptionYears, h_OptionYears, OPT_SZ, hipMemcpyHostToDevice);
  printf("Data init done.\n\n");

  printf("Executing Black-Scholes GPU kernel (%i iterations)...\n",
         NUM_ITERATIONS);
  int i;
  for (i = 0; i < NUM_ITERATIONS; i++) {
    BlackScholesGPU<<<DIV_UP((OPT_N / 2), 128), 128 /*480, 128*/>>>(
        (float2 *)d_CallResult, (float2 *)d_PutResult, (float2 *)d_StockPrice,
        (float2 *)d_OptionStrike, (float2 *)d_OptionYears, RISKFREE, VOLATILITY,
        OPT_N);
  }

  // Both call and put is calculated

  printf("\nReading back GPU results...\n");
  // Read back GPU results to compare them to CPU results
  hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost);
  hipMemcpy(h_PutResultGPU, d_PutResult, OPT_SZ, hipMemcpyDeviceToHost);

  printf("...releasing GPU memory.\n");
  hipFree(d_OptionYears);
  hipFree(d_OptionStrike);
  hipFree(d_StockPrice);
  hipFree(d_PutResult);
  hipFree(d_CallResult);
}

int main(int argc, char **argv) {
  // Start logs
  printf("[%s] - Starting...\n", argv[0]);

  //'h_' prefix - CPU (host) memory space
  float
      // Results calculated by CPU for reference
      *h_CallResultCPU,
      *h_PutResultCPU,
      // CPU copy of GPU results
      *h_CallResultGPU, *h_PutResultGPU,
      // CPU instance of input data
      *h_StockPrice, *h_OptionStrike, *h_OptionYears;

  double delta, ref, sum_delta, sum_ref, max_delta, L1norm, gpuTime;

  StopWatchInterface *hTimer = NULL;
  int i;

  findCudaDevice(argc, (const char **)argv);

  sdkCreateTimer(&hTimer);

  printf("Initializing data...\n");
  printf("...allocating CPU memory for options.\n");
  h_CallResultCPU = (float *)malloc(OPT_SZ);
  h_PutResultCPU = (float *)malloc(OPT_SZ);
  h_CallResultGPU = (float *)malloc(OPT_SZ);
  h_PutResultGPU = (float *)malloc(OPT_SZ);
  h_StockPrice = (float *)malloc(OPT_SZ);
  h_OptionStrike = (float *)malloc(OPT_SZ);
  h_OptionYears = (float *)malloc(OPT_SZ);

  printf("...generating input data in CPU mem.\n");
  srand(5347);

  // Generate options set
  for (i = 0; i < OPT_N; i++) {
    h_CallResultCPU[i] = 0.0f;
    h_PutResultCPU[i] = -1.0f;
    h_StockPrice[i] = RandFloat(5.0f, 30.0f);
    h_OptionStrike[i] = RandFloat(1.0f, 100.0f);
    h_OptionYears[i] = RandFloat(0.25f, 10.0f);
  }

  /*******************************************************************************/

  // Compute gradients
  auto callGrad = clad::gradient(
      launch, "h_CallResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears");
  auto putGrad = clad::gradient(
      launch, "h_PutResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears");

  // Declare and initialize the derivatives
  float* d_CallResultGPU = (float*)malloc(OPT_SZ);
  float* d_PutResultGPU = (float*)malloc(OPT_SZ);
  float* d_StockPrice = (float*)calloc(OPT_N, sizeof(float));
  float* d_OptionStrike = (float*)calloc(OPT_N, sizeof(float));
  float* d_OptionYears = (float*)calloc(OPT_N, sizeof(float));

  for (int i = 0; i < OPT_N; i++) {
    d_CallResultGPU[i] = 1.0f;
    d_PutResultGPU[i] = 1.0f;
  }

  /*******************************************************************************/

  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);
  // Compute the values and derivatives of the price of the call options
  callGrad.execute(h_CallResultCPU, h_CallResultGPU, h_PutResultCPU,
                   h_PutResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears,
                   d_CallResultGPU, d_StockPrice, d_OptionStrike,
                   d_OptionYears);

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  gpuTime = sdkGetTimerValue(&hTimer) / NUM_ITERATIONS;

  // Both call and put is calculated
  printf("Options count             : %i     \n", 2 * OPT_N);
  printf("BlackScholesGPU() time    : %f msec\n", gpuTime);
  printf("Effective memory bandwidth: %f GB/s\n",
         ((double)(5 * OPT_N * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
  printf("Gigaoptions per second    : %f     \n\n",
         ((double)(2 * OPT_N) * 1E-9) / (gpuTime * 1E-3));

  printf(
      "BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u "
      "options, NumDevsUsed = %u, Workgroup = %u\n",
      (((double)(2.0 * OPT_N) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime * 1e-3,
      (2 * OPT_N), 1, 128);

  printf("Checking the results...\n");
  printf("...running CPU calculations.\n\n");
  // Calculate options values on CPU
  BlackScholesCPU(h_CallResultCPU, h_PutResultCPU, h_StockPrice, h_OptionStrike,
                  h_OptionYears, RISKFREE, VOLATILITY, OPT_N);

  printf("Comparing the results...\n");
  // Calculate max absolute difference and L1 distance
  // between CPU and GPU results
  sum_delta = 0;
  sum_ref = 0;
  max_delta = 0;

  for (i = 0; i < OPT_N; i++) {
    ref = h_CallResultCPU[i];
    delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);

    if (delta > max_delta) {
      max_delta = delta;
    }

    sum_delta += delta;
    sum_ref += fabs(ref);
  }

  L1norm = sum_delta / sum_ref;
  printf("L1 norm: %E\n", L1norm);
  printf("Max absolute error: %E\n\n", max_delta);

  // Verify delta
  computeL1norm<Call, Delta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                             d_StockPrice);
  // Verify derivatives with respect to the Strike price
  computeL1norm<Call, dX>(h_StockPrice, h_OptionStrike, h_OptionYears,
                          d_OptionStrike);
  // Verify theta
  computeL1norm<Call, Theta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                             d_OptionYears);
  /*******************************************************************************/
  // Re-initialize data for next gradient call
  for (int i = 0; i < OPT_N; i++)
  {
      h_CallResultCPU[i] = 0.0f;
      h_PutResultCPU[i] = -1.0f;
      d_CallResultGPU[i] = 1.0f;
      d_PutResultGPU[i] = 1.0f;
  }
  for (int i = 0; i < OPT_N; i++)
  {
      d_StockPrice[i] = 0.f;
      d_OptionStrike[i] = 0.f;
      d_OptionYears[i] = 0.f;
  }
  // Compute the values and derivatives of the price of the Put options
  putGrad.execute(h_CallResultCPU, h_CallResultGPU, h_PutResultCPU,
                  h_PutResultGPU, h_StockPrice, h_OptionStrike, h_OptionYears,
                  d_PutResultGPU, d_StockPrice, d_OptionStrike, d_OptionYears);
  // Verify delta
  computeL1norm<Put, Delta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                            d_StockPrice);
  // Verify derivatives with respect to the Strike price
  computeL1norm<Put, dX>(h_StockPrice, h_OptionStrike, h_OptionYears,
                         d_OptionStrike);
  // Verify theta
  computeL1norm<Put, Theta>(h_StockPrice, h_OptionStrike, h_OptionYears,
                            d_OptionYears);
  /*******************************************************************************/

  printf("Shutting down...\n");
  printf("...releasing CPU memory.\n");
  free(h_OptionYears);
  free(h_OptionStrike);
  free(h_StockPrice);
  free(h_PutResultGPU);
  free(h_CallResultGPU);
  free(h_PutResultCPU);
  free(h_CallResultCPU);
  free(d_OptionYears);
  free(d_OptionStrike);
  free(d_StockPrice);
  free(d_PutResultGPU);
  free(d_CallResultGPU);
  sdkDeleteTimer(&hTimer);
  printf("Shutdown done.\n");

  printf("\n[BlackScholes] - Test Summary\n");

  if (L1norm > 1e-6) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n\n");
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}